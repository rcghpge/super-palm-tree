#include "hip/hip_runtime.h"
// Build and run
// nvcc -O3 -arch=native -o gpu_test gpu_test.cu
// ./gpu_test.cu

#include <cstdio>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(cmd) do { \
  hipError_t e = (cmd); \
  if (e != hipSuccess) { \
    fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
    exit(EXIT_FAILURE); \
  } \
} while (0)

__global__ void saxpy(const float a, const float* __restrict__ x, float* __restrict__ y, size_t n) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a * x[i] + y[i];
}

__global__ void device_memcopy(const float* __restrict__ src, float* __restrict__ dst, size_t n) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) dst[i] = src[i];
}

int main() {
  int count = 0;
  CHECK_CUDA(hipGetDeviceCount(&count));
  if (count == 0) { printf("No CUDA devices found.\n"); return 0; }

  int dev = 0;
  CHECK_CUDA(hipSetDevice(dev));
  hipDeviceProp_t prop{};
  CHECK_CUDA(hipGetDeviceProperties(&prop, dev));

  printf("CUDA Device 0: %s\n", prop.name);
  printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
  printf("  Global Memory: %.2f GB\n", prop.totalGlobalMem / (1024.0*1024.0*1024.0));
  printf("  Memory Bus Width: %d-bit\n", prop.memoryBusWidth);
  printf("  Memory Clock Rate: %.2f GHz (effective: vendor-specific)\n", prop.memoryClockRate/1e6);
  printf("  Multiprocessors: %d\n\n", prop.multiProcessorCount);

  // Problem size (~64 MB per vector)
  const size_t N = 1ull << 24;  // 16,777,216
  const size_t BYTES = N * sizeof(float);

  // Host buffers
  float *h_x=nullptr, *h_y=nullptr;
  CHECK_CUDA(hipHostMalloc(&h_x, BYTES));
  CHECK_CUDA(hipHostMalloc(&h_y, BYTES));
  for (size_t i=0; i<N; ++i) { h_x[i] = 1.0f; h_y[i] = 2.0f; }

  // Device buffers
  float *d_x=nullptr, *d_y=nullptr, *d_z=nullptr;
  CHECK_CUDA(hipMalloc(&d_x, BYTES));
  CHECK_CUDA(hipMalloc(&d_y, BYTES));
  CHECK_CUDA(hipMalloc(&d_z, BYTES));

  hipEvent_t start, stop;
  CHECK_CUDA(hipEventCreate(&start));
  CHECK_CUDA(hipEventCreate(&stop));

  // H2D bandwidth
  CHECK_CUDA(hipEventRecord(start));
  CHECK_CUDA(hipMemcpy(d_x, h_x, BYTES, hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_y, h_y, BYTES, hipMemcpyHostToDevice));
  CHECK_CUDA(hipEventRecord(stop));
  CHECK_CUDA(hipEventSynchronize(stop));
  float ms_h2d=0.f; CHECK_CUDA(hipEventElapsedTime(&ms_h2d, start, stop));
  double gb_h2d = (2.0 * BYTES) / 1e9;
  printf("H2D memcpy:  %.2f GB in %.3f ms  =>  %.2f GB/s\n", gb_h2d, ms_h2d, gb_h2d / (ms_h2d/1e3));

  // Device memory bandwidth - memcopy kernel
  const int block=256;
  const int grid = (int)((N + block - 1)/block);
  CHECK_CUDA(hipEventRecord(start));
  device_memcopy<<<grid, block>>>(d_x, d_z, N);
  CHECK_CUDA(hipEventRecord(stop));
  CHECK_CUDA(hipEventSynchronize(stop));
  float ms_dev=0.f; CHECK_CUDA(hipEventElapsedTime(&ms_dev, start, stop));

  // Reads + writes = 2 * BYTES
  double gb_dev = (2.0 * BYTES) / 1e9;
  printf("Device memcopy kernel: %.2f GB in %.3f ms  =>  %.2f GB/s\n", gb_dev, ms_dev, gb_dev / (ms_dev/1e3));

  // SAXPY compute ~2 flops/element
  const float a = 3.14159f;
  CHECK_CUDA(hipEventRecord(start));
  saxpy<<<grid, block>>>(a, d_x, d_y, N);
  CHECK_CUDA(hipEventRecord(stop));
  CHECK_CUDA(hipEventSynchronize(stop));
  float ms_saxpy=0.f; CHECK_CUDA(hipEventElapsedTime(&ms_saxpy, start, stop));
  double gflops = (2.0 * N) / 1e9 / (ms_saxpy/1e3);
  printf("SAXPY kernel: N=%zu in %.3f ms  =>  %.2f GFLOP/s\n", N, ms_saxpy, gflops);

  // D2H bandwidth
  CHECK_CUDA(hipEventRecord(start));
  CHECK_CUDA(hipMemcpy(h_y, d_y, BYTES, hipMemcpyDeviceToHost));
  CHECK_CUDA(hipEventRecord(stop));
  CHECK_CUDA(hipEventSynchronize(stop));
  float ms_d2h=0.f; CHECK_CUDA(hipEventElapsedTime(&ms_d2h, start, stop));
  double gb_d2h = BYTES / 1e9;
  printf("D2H memcpy:  %.2f GB in %.3f ms  =>  %.2f GB/s\n", gb_d2h, ms_d2h, gb_d2h / (ms_d2h/1e3));

  // Correctness check
  bool ok = true;
  for (size_t i=0; i<10; ++i) {
    if (h_y[i] != a*1.0f + 2.0f) { ok = false; break; }
  }
  printf("\nCorrectness: %s\n", ok ? "OK" : "FAILED");

  // Cleanup
  hipEventDestroy(start); hipEventDestroy(stop);
  hipFree(d_x); hipFree(d_y); hipFree(d_z);
  hipHostFree(h_x); hipHostFree(h_y);
  return ok ? 0 : 1;
}
